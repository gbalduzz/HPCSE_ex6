#include <cmath>
#include "cuda_potential.hpp"
#include "cuda_vector.hpp"
#include "timing.h"
#include "cuda_kernels.cuh"
#include "p2e.h"
#include <iostream>
using std::cout; using std::endl;
constexpr int threadsPerBlock = 128;




void cudaPotential(const Particles& p, Particles& t, const int order){
  const int Np= p.N;
  const int Nt= t.N;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);  
 
  reset_and_start_timer();
  // start copying while compuing p2e
  CudaVector<double> dtx(Nt,t.x),dty(Nt,t.y);
  CudaVector<double> dtw(Nt);
  std::vector<double> crd(order+1,0),cid(order+1,0);
  p2e(p, crd,cid);
  CudaVector<double> cr(order+1,crd.data()),ci(order+1,cid.data());
//time e2p
  hipEventRecord(start);
  e2p<<<Np/threadsPerBlock,threadsPerBlock>>>(dtx,dty,dtw,cr,ci,Nt);
  hipEventRecord(stop);
  dtw.copyTo(t.w);
  hipEventSynchronize(stop);
  double tf = get_elapsed_mcycles();


  cout<<"CUDA total M cycles (CPU p2e + GPU e2p) : "<<tf<<endl;
  float GPU_ms; hipEventElapsedTime(&GPU_ms, start, stop );
  cout<<"Time for e2p kernel: "<<GPU_ms<<endl;
}
 
