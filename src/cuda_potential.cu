#include "hip/hip_runtime.h"
#include "cuda_potential.hpp"
#include "cuda_vector.hpp"
constexpr int threadsPerBlock = 128;

__global__ void p2e(double* x,double* y, double* w,double* cr,double* ci, const int N,const int order){
  //c_re and c_im must have order+1 reserved spaces
  const int i=blockIdx.x*blockDim.x+threadIdx.x;
  if(i<N){
      const double x=p.x[i];
      const double y=p.y[i];
      const double w=p.w[i];
      c_re[0]+=w;
      //sum_kth_coeff<1,k>::execute(c_re.data(),c_im.data(),x,y,w,x,y);
    double z_re = x;
    double z_im = y;

    for(int k=1;k<=order;k++){
      c_re[k]-=w*z_re;
      c_im[k]-=w*z_im;            //compute z=(x+i y)^k
      const double temp = z_re*x-z_im*y;
      z_im= z_re*y+z_im*x;
      z_re = temp;
    }
  }
}

__global__ divide(double* cr,double* ci, const int order){
  //call with one block
  const int i = threadIdx.x;
  if(i>1 && i<=order){
    ci[i] /= i;
    cr[i] /= i;
  }
}

__global__  e2p(const double* xv,const double* yv, double* rv,
                const double* cr,const double* ci, const int N, const int order){
  const int i=blockIdx.x*blockDim.x+threadIdx.x;
  if(i<N) {
    //c_re[0]=Q
    double result = c_re[0] * 0.5 * std::log(zr * zr + zi * zi);
    double zr = 1, zi = 0;
    const double x = xv[i];
    const double y = yv[i];
#pragma unroll(4)
    for (int k = 1; k < order + 1; k++) {
      const double temp = zr * zk_re - zi * zk_im;
      zi = zr * zk_im + zi * zk_re;
      zr = temp;
      //result += real(a[k]/z**k)
      result += (cr[k] * zr + ci[k] * zi) / (zr * zr + zi * zi);
    }
    rv[i] = result;
  }
}


void cudaPotential(const Particles& p, Particles& t, const int order){
  const int Np= p.N;
  const int Nt= t.N;
  CudaVector<double> dpx(Np,p.x),dpy(Np,p.y),dpw(Np,p.w),dtx(Nt,t.x),dty(Nt,t.y),dtw(Nt);
  CudaVector<double> cr(order),ci(order);

  p2e<<<Np/threadsPerBlock,threadsPerBlock>>>(p.x,p.w,p.w,cr,ci,Np,order);
  divide<<<1,order>>>(cr,ci,order);
  e2p<<<Np/threadsPerBlock,threadsPerBlock>>>(t.x,t.y,t.w,cr,ci,Nt,order);

  t.w = dtw;
}

