#include "hip/hip_runtime.h"
#pragma once
#include "cuda_vector.hpp"
constexpr int threadsPerBlock = 128;

__global__ void p2e(double* x,double* y, double* w,double* cr,double* ci, const int N,const int order){
  //c_re and c_im must have order+1 reserved spaces
  const int i=blockIdx.x*blockDim.x+threadIdx.x;
  if(i<N){
      const double x=p.x[i];
      const double y=p.y[i];
      const double w=p.w[i];
      c_re[0]+=w;
      //sum_kth_coeff<1,k>::execute(c_re.data(),c_im.data(),x,y,w,x,y);
    double z_re = x;
    double z_im = y;

    for(int k=1;k<=order;k++){
      c_re[k]-=w*z_re;
      c_im[k]-=w*z_im;            //compute z=(x+i y)^k
      const double temp = z_re*x-z_im*y;
      z_im= z_re*y+z_im*x;
      z_re = temp;
    }
  }
}

void cudaPotential(const Particles& p, Particles& t, const int order){
  const int Np= p.N;
  const int Nt= t.N;
  CudaVector<double> dpx(Np,p.x),dpy(Np,p.y),dpw(Np,p.w),dtx(Nt,t.x),dty(Nt,t.y),dtw(Nt);
  CudaVector<double> cr(order),ci(order);
//do a bunch of stuff
  //p2e
  p2e<<<Np/threadsPerBlock,threadsPerBlock>>>(p.x,p.w,p.w,cr,ci,Np,order);
  // for(int i=2;i<k+1;i++) {c_re[i]/=i;c_im[i]/=i;}
  //e2p
  t.w = dtw;
}

