#include "hip/hip_runtime.h"
#include <cmath>
#include "cuda_potential.hpp"
#include "cuda_vector.hpp"
constexpr int threadsPerBlock = 128;

__global__ void p2e(double* xv,double* yv, double* wv,double* cr,double* ci, const int N,const int order){
  //cr and ci must have order+1 reserved spaces
  const int i=blockIdx.x*blockDim.x+threadIdx.x;
  if(i<N){
      const double x=xv[i];
      const double y=yv[i];
      const double w=wv[i];
      cr[0]+=w;
      double z_re = x;
      double z_im = y;

    for(int k=1;k<=order;k++){
      cr[k]-=w*z_re;
      ci[k]-=w*z_im;
      //compute z=(x+i y)^k
      const double temp = z_re*x-z_im*y;
      z_im= z_re*y+z_im*x;
      z_re = temp;
    }
  }
}

__global__ void divide(double* cr,double* ci, const int order){
  //call with one block
  const int i = threadIdx.x;
  if(i>1 && i<=order){
    ci[i] /= i;
    cr[i] /= i;
  }
}

__global__  void e2p(const double* xv,const double* yv, double* rv,
                const double* cr,const double* ci, const int N, const int order){
  const int i=blockIdx.x*blockDim.x+threadIdx.x;
  if(i<N) {
    //cr[0]=Q
    const double x = xv[i];		   
    const double y = yv[i];
    double result = cr[0] * 0.5 * std::log(x * x + y * y);
    double zr = 1, zi = 0;
   
//#pragma unroll(4)
    for (int k = 1; k < order + 1; k++) {
      const double temp = zr * x - zi * y;
      zi = zr * y + zi * x;
      zr = temp;
      //result += real(a[k]/z**k)
      result += (cr[k] * zr + ci[k] * zi) / (zr * zr + zi * zi);
    }
    rv[i] = result;
  }
}


void cudaPotential(const Particles& p, Particles& t, const int order){
  const int Np= p.N;
  const int Nt= t.N;
  CudaVector<double> dpx(Np,p.x),dpy(Np,p.y),dpw(Np,p.w);
  CudaVector<double> dtx(Nt,t.x),dty(Nt,t.y);
  CudaVector<double> dtw(Nt);
  // are cr,ci set to zero?
  CudaVector<double> cr(order+1),ci(order+1);

  p2e<<<Np/threadsPerBlock,threadsPerBlock>>>(p.x,p.w,p.w,cr,ci,Np,order);
  divide<<<1,order+1>>>(cr,ci,order);
  e2p<<<Np/threadsPerBlock,threadsPerBlock>>>(t.x,t.y,t.w,cr,ci,Nt,order);

  dtw.copyTo(t.w);
}
 
