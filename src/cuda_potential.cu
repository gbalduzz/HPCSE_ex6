#include "hip/hip_runtime.h"
#include <cmath>
#include "cuda_potential.hpp"
#include "cuda_vector.hpp"
#include "cuda_kernels.cuh"
constexpr int threadsPerBlock = 128;


__global__ void divide(double* cr,double* ci, const int order){
  //call with one block
  const int i = threadIdx.x;
  if(i>1 && i<=order){
    ci[i] /= i;
    cr[i] /= i;
  }
}


void cudaPotential(const Particles& p, Particles& t, const int order){
  const int Np= p.N;
  const int Nt= t.N;
  CudaVector<double> dpx(Np,p.x),dpy(Np,p.y),dpw(Np,p.w);
  CudaVector<double> dtx(Nt,t.x),dty(Nt,t.y);
  CudaVector<double> dtw(Nt);
  // are cr,ci set to zero?
  CudaVector<double> cr(order+1),ci(order+1);

  p2e<<<Np/threadsPerBlock,threadsPerBlock>>>(p.x,p.w,p.w,cr,ci,Np);
  divide<<<1,order+1>>>(cr,ci,order);
  e2p<<<Np/threadsPerBlock,threadsPerBlock>>>(t.x,t.y,t.w,cr,ci,Nt);

  dtw.copyTo(t.w);
}
 
